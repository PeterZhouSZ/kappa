#include "hip/hip_runtime.h"
#include <kappa/core.hpp>


__global__
void raycast_volume_kernel(
    volume<voxel> vol,
    image<float3> vm,
    image<float4> nm,
    intrinsics K,
    mat4x4 T,
    float mu,
    float near,
    float far)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;
    if (u >= K.width || v >= K.height) return;

    float3 q;
    q.x = (u - K.cx) / K.fx;
    q.y = (v - K.cy) / K.fy;
    q.z = 1.0f;

    float3 origin = {T.m03, T.m13, T.m23};
    float3 direction = rotate(T, q);

    float z = near;
    float3 p = origin + direction * z;

    float ft = vol.nearest(p);
    float ftt;
    float step = 0.8f * mu;
    float min_step = vol.voxel_size;
    for (; z <= far; z += step) {
        p = origin + direction * z;
        ftt = vol.interp(p);
        if (ftt < 0.0f) break;
        step = fmaxf(0.8f * ftt * mu, min_step);
        ft = ftt;
    }

    if (ftt < 0.0f) z += step * ftt / (ft - ftt);
    else z = -1.0f;

    int i = u + v * K.width;
    vm[i] = {0.0f, 0.0f, 0.0f};
    nm[i] = {0.0f, 0.0f, 0.0f, 0.0f};
    if (z >= 0.0f) {
        p = origin + direction * z;
        vm[i] = p;
        nm[i] = make_float4(vol.grad(p));
    }
}


__global__
void raycast_z_buffer_kernel(
    cloud<surfel> pcd,
    image<uint32_t> zbuf,
    intrinsics K,
    mat4x4 T,
    float maxw)
{
    int k = threadIdx.x + blockIdx.x * blockDim.x;
    if (k >= pcd.size) return;

    float3 p = pcd[k].pos;
    float3 q = T * p;
    if (q.z <= 0.001f) return;

    int u = roundf((q.x / q.z) * K.fx + K.cx);
    int v = roundf((q.y / q.z) * K.fy + K.cy);
    if (u < 0 || u >= K.width || v < 0 || v >= K.height) return;

    float f = 0.5 * (K.fx + K.fy);
    int r = pcd[k].radius * f / q.z + 0.5f;
    int r2 = r * r; // radius squared

    bool unstable = (pcd[k].weight < maxw);
    uint32_t z = q.z * ZBUFFER_SCALE + unstable * Z_OFFSET;
    for (int dy = -r; dy <= r; ++dy) {
        for (int dx = -r; dx <= r; ++dx) {
            int x = u + dx;
            int y = v + dy;
            if (x < 0 || x >= K.width || y < 0 || y >= K.height) continue;
            if (dx * dx + dy * dy > r2) continue;

            int i = x + y * K.width;
            atomicMin(&zbuf[i], z);
        }
    }
}


__global__
void raycast_index_kernel(
    cloud<surfel> pcd,
    image<uint32_t> zbuf,
    image<uint32_t> idm,
    intrinsics K,
    mat4x4 T,
    float maxw)
{
    int k = threadIdx.x + blockIdx.x * blockDim.x;
    if (k >= pcd.size) return;

    float3 p = pcd[k].pos;
    float3 q = T * p;
    if (q.z <= 0.001f) return;

    int u = roundf((q.x / q.z) * K.fx + K.cx);
    int v = roundf((q.y / q.z) * K.fy + K.cy);
    if (u < 0 || u >= K.width || v < 0 || v >= K.height) return;

    float f = 0.5 * (K.fx + K.fy);
    int r = pcd[k].radius * f / q.z + 0.5f;
    int r2 = r * r; // radius squared

    bool unstable = (pcd[k].weight < maxw);
    uint32_t z = q.z * ZBUFFER_SCALE + unstable * Z_OFFSET;
    for (int dy = -r; dy <= r; ++dy) {
        for (int dx = -r; dx <= r; ++dx) {
            int x = u + dx;
            int y = v + dy;
            if (x < 0 || x >= K.width || y < 0 || y >= K.height) continue;
            if (dx * dx + dy * dy > r2) continue;

            int i = x + y * K.width;
            if (z > zbuf[i]) continue;
            idm[i] = k + 1;
        }
    }
}


__global__
void raycast_cloud_kernel(
    cloud<surfel> pcd,
    image<float3> vm,
    image<float4> nm,
    image<float3> cm,
    image<uint32_t> idm,
    intrinsics K,
    int timestamp,
    float maxw,
    float cutoff)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;
    if (u >= K.width || v >= K.height) return;

    int i = u + v * K.width;
    vm[i] = {0.0f, 0.0f, 0.0f};
    nm[i] = {0.0f, 0.0f, 0.0f};
    cm[i] = {0.0f, 0.0f, 0.0f};

    int k = idm[i] - 1;
    if (k < 0) return;
    if (pcd[k].pos.z > cutoff) return;
    if (pcd[k].weight < maxw * 0.5f) return;

    float3 n = pcd[k].normal;
    vm[i] = pcd[k].pos;
    nm[i] = {n.x, n.y, n.z, pcd[k].radius};
    cm[i] = pcd[k].color;
}


void raycast_volume(const volume<voxel> vol,
                    image<float3>* vm,
                    image<float4>* nm,
                    intrinsics K,
                    mat4x4 T,
                    float mu,
                    float near,
                    float far)
{
    dim3 block_size(16, 16);
    dim3 grid_size;
    grid_size.x = divup(K.width, block_size.x);
    grid_size.y = divup(K.height, block_size.y);
    raycast_volume_kernel<<<grid_size, block_size>>>(
        vol.cuda(), vm->cuda(), nm->cuda(), K, T, mu, near, far);
}


void raycast_cloud(const cloud<surfel> pcd,
                   image<float3>* vm,
                   image<float4>* nm,
                   image<float3>* cm,
                   image<uint32_t>* idm,
                   intrinsics K,
                   mat4x4 T,
                   int timestamp,
                   float maxw,
                   float cutoff)
{
    static image<uint32_t> zbuf;
    zbuf.resize(K.width, K.height, DEVICE_CUDA);
    zbuf.clear(0xff);
    idm->clear();
    {
        uint32_t block_size = 512;
        uint32_t grid_size = divup(pcd.size, block_size);
        raycast_z_buffer_kernel<<<grid_size, block_size>>>(
            pcd.cuda(), zbuf.cuda(), K, T.inverse(), maxw);
        raycast_index_kernel<<<grid_size, block_size>>>(
            pcd.cuda(), zbuf.cuda(), idm->cuda(), K, T.inverse(), maxw);
    }
    {
        dim3 block_size(16, 16);
        dim3 grid_size;
        grid_size.x = divup(K.width, block_size.x);
        grid_size.y = divup(K.height, block_size.y);
        raycast_cloud_kernel<<<grid_size, block_size>>>(
            pcd.cuda(), vm->cuda(), nm->cuda(), cm->cuda(),
            idm->cuda(), K, timestamp, maxw, cutoff);
    }
}
