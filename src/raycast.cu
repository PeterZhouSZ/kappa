#include "hip/hip_runtime.h"
#include <kappa/core.hpp>


__global__
void raycast_volume_kernel(
    volume<voxel> vol,
    image<float3> vm,
    image<float4> nm,
    intrinsics K,
    mat4x4 T,
    float mu,
    float near,
    float far)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;
    if (u >= K.width || v >= K.height) return;

    float3 q;
    q.x = (u - K.cx) / K.fx;
    q.y = (v - K.cy) / K.fy;
    q.z = 1.0f;

    float3 origin = {T.m03, T.m13, T.m23};
    float3 direction = rotate(T, q);

    float z = near;
    float3 p = origin + direction * z;

    float ft = vol.nearest(p);
    float ftt;
    float step = 0.8f * mu;
    float min_step = vol.voxel_size;
    for (; z <= far; z += step) {
        p = origin + direction * z;
        ftt = vol.interp(p);
        if (ftt < 0.0f) break;
        step = fmaxf(0.8f * ftt * mu, min_step);
        ft = ftt;
    }

    if (ftt < 0.0f) z += step * ftt / (ft - ftt);
    else z = -1.0f;

    int i = u + v * K.width;
    vm[i] = {0.0f, 0.0f, 0.0f};
    nm[i] = {0.0f, 0.0f, 0.0f, 0.0f};
    if (z >= 0.0f) {
        p = origin + direction * z;
        vm[i] = p;
        nm[i] = make_float4(vol.grad(p));
    }
}


__global__
void raycast_z_buffer_kernel(
    cloud<surfel> pcd,
    image<uint32_t> zbuf,
    intrinsics K,
    mat4x4 T)
{
    int k = threadIdx.x + blockIdx.x * blockDim.x;
    if (k >= pcd.size) return;

    float3 p = pcd[k].pos;
    float3 q = T * p;
    if (q.z <= 0.001f) return;

    int u = roundf((q.x / q.z) * K.fx + K.cx);
    int v = roundf((q.y / q.z) * K.fy + K.cy);
    if (u < 0 || u >= K.width || v < 0 || v >= K.height) return;

    float f = 0.5 * (K.fx + K.fy);
    int r = pcd[k].radius * f / q.z + 0.5f;
    int r2 = r * r; // radius squared

    uint32_t z = q.z * ZBUFFER_SCALE;
    for (int dy = -r; dy <= r; ++dy) {
        for (int dx = -r; dx <= r; ++dx) {
            int x = u + dx;
            int y = v + dy;
            if (x < 0 || x >= K.width || y < 0 || y >= K.height) continue;
            if (dx * dx + dy * dy > r2) continue;

            int i = x + y * K.width;
            atomicMin(&zbuf[i], z);
        }
    }
}


__global__
void raycast_index_kernel(
    cloud<surfel> pcd,
    image<uint32_t> zbuf,
    image<uint32_t> idm,
    intrinsics K,
    mat4x4 T)
{
    int k = threadIdx.x + blockIdx.x * blockDim.x;
    if (k >= pcd.size) return;

    float3 p = pcd[k].pos;
    float3 q = T * p;
    if (q.z <= 0.001f) return;

    int u = roundf((q.x / q.z) * K.fx + K.cx);
    int v = roundf((q.y / q.z) * K.fy + K.cy);
    if (u < 0 || u >= K.width || v < 0 || v >= K.height) return;

    float f = 0.5 * (K.fx + K.fy);
    int r = pcd[k].radius * f / q.z + 0.5f;
    int r2 = r * r; // radius squared

    uint32_t z = q.z * ZBUFFER_SCALE;
    for (int dy = -r; dy <= r; ++dy) {
        for (int dx = -r; dx <= r; ++dx) {
            int x = u + dx;
            int y = v + dy;
            if (x < 0 || x >= K.width || y < 0 || y >= K.height) continue;
            if (dx * dx + dy * dy > r2) continue;

            int i = x + y * K.width;
            if (z > zbuf[i]) continue;
            idm[i] = k + 1;
        }
    }
}


__global__
void raycast_cloud_kernel(
    cloud<surfel> pcd,
    image<uint32_t> idm,
    image<float3> vm,
    image<float4> nm,
    intrinsics K)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;
    if (u >= K.width || v >= K.height) return;

    int i = u + v * K.width;
    vm[i] = {0.0f, 0.0f, 0.0f};
    nm[i] = {0.0f, 0.0f, 0.0f};

    int k = idm[i];
    if (k == 0) return;

    vm[i] = pcd[k - 1].pos;
    nm[i] = make_float4(pcd[k - 1].normal);
}


void raycast(const volume<voxel> vol,
             image<float3>* vm,
             image<float4>* nm,
             intrinsics K,
             mat4x4 T,
             float mu,
             float near,
             float far)
{
    dim3 block_size(16, 16);
    dim3 grid_size;
    grid_size.x = divup(K.width, block_size.x);
    grid_size.y = divup(K.height, block_size.y);
    raycast_volume_kernel<<<grid_size, block_size>>>(
        vol.cuda(), vm->cuda(), nm->cuda(), K, T, mu, near, far);
}


void raycast(const cloud<surfel> pcd,
             image<float3>* vm,
             image<float4>* nm,
             image<uint32_t>* idm,
             intrinsics K,
             mat4x4 T)
{
    static image<uint32_t> zbuf;
    zbuf.resize(K.width, K.height, DEVICE_CUDA);
    zbuf.clear(0xff);
    idm->clear();
    {
        unsigned int block_size = 512;
        unsigned int grid_size = divup(pcd.size, block_size);
        raycast_z_buffer_kernel<<<grid_size, block_size>>>(
            pcd.cuda(), zbuf.cuda(), K, T.inverse());
        raycast_index_kernel<<<grid_size, block_size>>>(
            pcd.cuda(), zbuf.cuda(), idm->cuda(), K, T.inverse());
    }
    {
        dim3 block_size(16, 16);
        dim3 grid_size;
        grid_size.x = divup(K.width, block_size.x);
        grid_size.y = divup(K.height, block_size.y);
        raycast_cloud_kernel<<<grid_size, block_size>>>(
            pcd.cuda(), idm->cuda(), vm->cuda(), nm->cuda(), K);
    }
}
