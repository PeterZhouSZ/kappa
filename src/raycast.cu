#include "hip/hip_runtime.h"
#include <kappa/core.hpp>
#define ZBUFFER_SCALE 100000


__global__
void raycast_volume_kernel(volume<sdf32f_t> vol, image<float3> vm, image<float3> nm, intrinsics K, mat4x4 T, float mu, float near, float far)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;
    if (u >= K.width || v >= K.height) return;

    float3 q;
    q.x = (u - K.cx) / K.fx;
    q.y = (v - K.cy) / K.fy;
    q.z = 1.0f;

    float3 origin = {T.m03, T.m13, T.m23};
    float3 direction = rotate(T, q);

    float z = near;
    float3 p = origin + direction * z;

    float ft = nearest_tsdf(vol, p);
    float ftt;
    float step = 0.8f * mu;
    float min_step = vol.voxel_size;
    for (; z <= far; z += step) {
        p = origin + direction * z;
        ftt = interp_tsdf(vol, p);
        if (ftt < 0.0f) break;
        step = fmaxf(0.8f * ftt * mu, min_step);
        ft = ftt;
    }

    if (ftt < 0.0f) z += step * ftt / (ft - ftt);
    else z = -1.0f;

    int i = u + v * K.width;
    vm.data[i] = {0.0f, 0.0f, 0.0f};
    nm.data[i] = {0.0f, 0.0f, 0.0f};
    if (z >= 0.0f) {
        p = origin + direction * z;
        vm.data[i] = p;
        nm.data[i] = grad_tsdf(vol, p);
    }
}


__global__
void update_z_buffer(cloud<surfel32f_t> pc, image<uint32_t> zbuf, intrinsics K, mat4x4 T)
{
    int k = threadIdx.x + blockIdx.x * blockDim.x;
    if (k >= pc.size) return;

    float3 p = pc.data[k].pos;
    float3 q = T * p;
    // if (q.z <= 0.001f) return;

    int u = roundf((q.x / q.z) * K.fx + K.cx);
    int v = roundf((q.y / q.z) * K.fy + K.cy);
    if (u < 0 || u >= K.width || v < 0 || v >= K.height) return;

    int i = u + v * K.width;
    uint32_t z = q.z * ZBUFFER_SCALE;
    atomicMin(&zbuf.data[i], z);
}


void raycast_volume(const volume<sdf32f_t>* vol, image<float3>* vm, image<float3>* nm, intrinsics K, mat4x4 T, float mu, float near, float far)
{
    dim3 block_size(16, 16);
    dim3 grid_size;
    grid_size.x = divup(K.width, block_size.x);
    grid_size.y = divup(K.height, block_size.y);
    raycast_volume_kernel<<<grid_size, block_size>>>(vol->gpu(), vm->gpu(), nm->gpu(), K, T, mu, near, far);
}


void raycast_cloud(const cloud<surfel32f_t>* pc, image<float3>* vm, image<float4>* nm, image<uint4>* im, intrinsics K, mat4x4 T)
{
    unsigned int block_size = 512;
    unsigned int grid_size = divup(pc->size, block_size);

    image<uint32_t> zbuf;
    zbuf.allocate(K.width, K.height, DEVICE_CUDA);
    zbuf.clear(0xff);

    update_z_buffer<<<grid_size, block_size>>>(pc->gpu(), zbuf.gpu(), K, T.inverse());

    zbuf.deallocate();
}
