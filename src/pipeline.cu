#include "hip/hip_runtime.h"
#include <kinfu/pipeline.hpp>
#include <kinfu/math.hpp>


namespace kinfu {

__global__
void compute_vertex_kernel(image<uint16_t> dmap, image<float3> vmap, intrinsics K, float cutoff)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;
    if (u >= K.width || v >= K.height) return;

    int i = u + v * K.width;
    float d = dmap.data[i] * 0.001f;
    if (d > cutoff) d = 0.0f;

    vmap.data[i].x = (u - K.cx) * d / K.fx;
    vmap.data[i].y = (v - K.cy) * d / K.fy;
    vmap.data[i].z = d;
}


__global__
void compute_normal_kernel(image<float3> vmap, image<float3> nmap, intrinsics K)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;
    if (u <= 0 || u >= K.width - 1 || v <= 0 || v >= K.height - 1) return;

    float3 v00 = vmap.data[(u - 1) + v * K.width];
    float3 v10 = vmap.data[(u + 1) + v * K.width];
    float3 v01 = vmap.data[u + (v - 1) * K.width];
    float3 v11 = vmap.data[u + (v + 1) * K.width];

    float3 normal = {0.0f, 0.0f, 0.0f};
    if (v00.z != 0 && v01.z != 0 && v10.z != 0 && v11.z != 0) {
        float3 dx = v00 - v10;
        float3 dy = v01 - v11;
        normal = normalize(cross(dy, dx));
    }
    nmap.data[u + v * K.width] = normal;
}


__global__
void integrate_volume_kernel(volume<sdf32f_t> vol, image<uint16_t> dmap, intrinsics K, mat4x4 P, float mu)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    if (x >= vol.dimension.x || y >= vol.dimension.y || z >= vol.dimension.z) return;

    float3 p = {(float)x, (float)y, (float)z};
    float3 q = vol.offset + p * vol.voxel_size;
    if (q.z <= 0.001f) return;

    int u = roundf((q.x / q.z) * K.fx + K.cx);
    int v = roundf((q.y / q.z) * K.fy + K.cy);
    if (u < 0 || u >= K.width || v < 0 || v >= K.height) return;

    float d = dmap.data[u + v * K.width] * 0.001f;
    if (d <= 0.0f) return;

    float dist = d - q.z;
    if (dist <= -mu) return;

    int i = x + y * vol.dimension.x + z * vol.dimension.x * vol.dimension.y;
    float ftt = fminf(1.0f, dist / mu);
    float wtt = 1.0f;
    float ft  = vol.data[i].tsdf;
    float wt  = vol.data[i].weight;
    vol.data[i].tsdf = (ft * wt + ftt * wtt) / (wt + wtt);
    vol.data[i].weight = wt + wtt;
}


__device__
float interp_tsdf(volume<sdf32f_t> vol, float3 p)
{
    int x = roundf((p.x - vol.offset.x) / vol.voxel_size);
    int y = roundf((p.y - vol.offset.y) / vol.voxel_size);
    int z = roundf((p.z - vol.offset.z) / vol.voxel_size);
    if (x < 0 || x >= vol.dimension.x ||
        y < 0 || y >= vol.dimension.y ||
        z < 0 || z >= vol.dimension.z)
        return 1.0f; // cannot interpolate
    int i = x + y * vol.dimension.x + z * vol.dimension.x * vol.dimension.y;
    return vol.data[i].tsdf;
}


__global__
void raycast_volume_kernel(volume<sdf32f_t> vol, image<float3> vmap, intrinsics K, mat4x4 P)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;
    if (u >= K.width || v >= K.height) return;

    const float near = 0.001f;
    const float far  = 4.0f;

    float3 q;
    q.x = (u - K.cx) / K.fx;
    q.y = (v - K.cy) / K.fy;
    q.z = 1.0f;

    float3 origin = {0.0f, 0.0f, 0.0f};
    float3 direction = q;

    float z = near;
    float3 p = origin + direction * z;

    float ft, ftt;
    float step = vol.voxel_size;
    for (; z <= far; z += step) {
        p = origin + direction * z;
        ftt = interp_tsdf(vol, p);
        if (ftt < 0.0f) break;
        ft = ftt;
    }

    if (ftt < 0.0f) z += step * ftt / (ft - ftt);
    else z = -1.0f;

    p = {0.0f, 0.0f, 0.0f};
    if (z > 0.0f) p = origin + direction * z;
    vmap.data[u + v * K.width] = p;
}


static void compute_vertex_map(const image<uint16_t>* dmap, image<float3>* vmap, intrinsics K, float cutoff)
{
    dim3 block_size(16, 16);
    dim3 grid_size;
    grid_size.x = divup(K.width, block_size.x);
    grid_size.y = divup(K.height, block_size.y);
    compute_vertex_kernel<<<grid_size, block_size>>>(dmap->gpu(), vmap->gpu(), K, cutoff);
}


static void compute_normal_map(const image<float3>* vmap, image<float3>* nmap, intrinsics K)
{
    dim3 block_size(16, 16);
    dim3 grid_size;
    grid_size.x = divup(K.width, block_size.x);
    grid_size.y = divup(K.height, block_size.y);
    compute_normal_kernel<<<grid_size, block_size>>>(vmap->gpu(), nmap->gpu(), K);
}


static void integrate_volume(const volume<sdf32f_t>* vol, image<uint16_t>* dmap, intrinsics K, mat4x4 P, float mu)
{
    dim3 block_size(8, 8, 8);
    dim3 grid_size;
    grid_size.x = divup(vol->dimension.x, block_size.x);
    grid_size.y = divup(vol->dimension.y, block_size.y);
    grid_size.z = divup(vol->dimension.z, block_size.z);
    integrate_volume_kernel<<<grid_size, block_size>>>(vol->gpu(), dmap->gpu(), K, P, mu);
}


static void raycast_volume(const volume<sdf32f_t>* vol, image<float3>* vmap, intrinsics K, mat4x4 P)
{
    dim3 block_size(16, 16);
    dim3 grid_size;
    grid_size.x = divup(K.width, block_size.x);
    grid_size.y = divup(K.height, block_size.y);
    raycast_volume_kernel<<<grid_size, block_size>>>(vol->gpu(), vmap->gpu(), K, P);
}


static void extract_isosurface_cloud(const volume<sdf32f_t>* vol, point_cloud* pc)
{
}


pipeline::pipeline()
{
    hipSetDeviceFlags(hipDeviceMapHost);
}


pipeline::~pipeline()
{
    dmap.deallocate();
    cmap.deallocate();
    vmap.deallocate();
    nmap.deallocate();
}


void pipeline::process()
{
    cam->read(&dmap, &cmap);
    preprocess();
    integrate();
    raycast();
}


void pipeline::preprocess()
{
    vmap.resize(dmap.width, dmap.height, ALLOCATOR_DEVICE);
    nmap.resize(dmap.width, dmap.height, ALLOCATOR_DEVICE);
    compute_vertex_map(&dmap, &vmap, cam->K, cutoff);
    compute_normal_map(&vmap, &nmap, cam->K);
}


void pipeline::integrate()
{
    integrate_volume(vol, &dmap, cam->K, P, mu);
}


void pipeline::raycast()
{
    rvmap.resize(vmap.width, vmap.height, ALLOCATOR_MAPPED);
    rnmap.resize(nmap.width, nmap.height, ALLOCATOR_MAPPED);
    raycast_volume(vol, &rvmap, cam->K, P);
}


void pipeline::extract_point_cloud(point_cloud* pc)
{
    extract_isosurface_cloud(vol, pc);
}

}
