#include "hip/hip_runtime.h"
#include <kappa/core.hpp>


__global__
void depth_bilateral_kernel(image<float> dm0, image<float> dm1, intrinsics K, float d_sigma, float r_sigma)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;
    if (u >= K.width || v >= K.height) return;

    int i = u + v * K.width;
    dm1.data[i] = 0.0f;
    float p = dm0.data[i];
    if (p == 0.0f) return;

    float sum = 0.0f;
    float count = 0.0f;
    float inv_r_sigma2 = -1.0f / (2.0f * r_sigma * r_sigma);
    float inv_d_sigma2 = -1.0f / (2.0f * d_sigma * d_sigma);

    int radius = 2;
    for (int dy = -radius; dy <= radius; ++dy) {
        for (int dx = -radius; dx <= radius; ++dx) {
            int x = u + dx;
            int y = v + dy;
            if (x < 0 || x >= K.width || y < 0 || y >= K.height) continue;

            float q = dm0.data[x + y * K.width];
            if (q == 0.0f) continue;

            float w_r = __expf(dx * dx * inv_r_sigma2) * __expf(dy * dy * inv_r_sigma2);
            float w_d = __expf((p - q) * (p - q) * inv_d_sigma2);
            sum += q * w_r * w_d;
            count += w_r * w_d;
        }
    }
    dm1.data[i] = (sum / count);
}


__global__
void compute_depth_kernel(image<uint16_t> rm, image<float> dm, intrinsics K, float cutoff)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;
    if (u >= K.width || v >= K.height) return;

    int i = u + v * K.width;
    float d = rm.data[i] * 0.001f;
    if (d > cutoff) d = 0.0f;
    dm.data[i] = d;
}


__global__
void compute_vertex_kernel(image<float> dm, image<float3> vm, intrinsics K)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;
    if (u >= K.width || v >= K.height) return;

    int i = u + v * K.width;
    float d = dm.data[i];

    vm.data[i].x = (u - K.cx) * d / K.fx;
    vm.data[i].y = (v - K.cy) * d / K.fy;
    vm.data[i].z = d;
}


__global__
void compute_normal_kernel(image<float3> vm, image<float3> nm, intrinsics K)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;
    if (u <= 0 || u >= K.width - 1 || v <= 0 || v >= K.height - 1) return;

    float3 v00 = vm.data[(u - 1) + v * K.width];
    float3 v10 = vm.data[(u + 1) + v * K.width];
    float3 v01 = vm.data[u + (v - 1) * K.width];
    float3 v11 = vm.data[u + (v + 1) * K.width];

    float3 normal = {0.0f, 0.0f, 0.0f};
    if (v00.z != 0 && v01.z != 0 && v10.z != 0 && v11.z != 0) {
        float3 dx = v00 - v10;
        float3 dy = v01 - v11;
        normal = normalize(cross(dy, dx));
    }
    nm.data[u + v * K.width] = normal;
}


__global__
void compute_normal_radius_kernel(image<float3> vm, image<float4> nm, intrinsics K)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;
    if (u <= 0 || u >= K.width - 1 || v <= 0 || v >= K.height - 1) return;

    int i = u + v * K.width;
    float3 v00 = vm.data[(u - 1) + v * K.width];
    float3 v10 = vm.data[(u + 1) + v * K.width];
    float3 v01 = vm.data[u + (v - 1) * K.width];
    float3 v11 = vm.data[u + (v + 1) * K.width];

    float3 normal = {0.0f, 0.0f, 0.0f};
    if (v00.z != 0 && v01.z != 0 && v10.z != 0 && v11.z != 0) {
        float3 dx = v00 - v10;
        float3 dy = v01 - v11;
        normal = normalize(cross(dy, dx));
    }

    float r = 0.0f;
    if (length(normal) > 0.0f) {
        float d = vm.data[i].z;
        float f = 0.5f * (K.fx + K.fy);
        r = sqrtf(2.0f) * d / f;
    }

    nm.data[i].x = normal.x;
    nm.data[i].y = normal.y;
    nm.data[i].z = normal.z;
    nm.data[i].w = r;
}


__global__
void reset_volume_kernel(volume<sdf32f_t> vol)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    if (x >= vol.dimension.x || y >= vol.dimension.y || z >= vol.dimension.z) return;
    int i = x + y * vol.dimension.x + z * vol.dimension.x * vol.dimension.y;
    vol.data[i].tsdf = 1.0f;
    vol.data[i].weight = 0.0f;
}


void compute_depth_map(const image<uint16_t>* rm, image<float>* dm, intrinsics K, float cutoff)
{
    dim3 block_size(16, 16);
    dim3 grid_size;
    grid_size.x = divup(K.width, block_size.x);
    grid_size.y = divup(K.height, block_size.y);
    compute_depth_kernel<<<grid_size, block_size>>>(rm->gpu(), dm->gpu(), K, cutoff);
}


void depth_bilateral(const image<float>* dm0, image<float>* dm1, intrinsics K, float d_sigma, float r_sigma)
{
    dim3 block_size(16, 16);
    dim3 grid_size;
    grid_size.x = divup(K.width, block_size.x);
    grid_size.y = divup(K.height, block_size.y);
    depth_bilateral_kernel<<<grid_size, block_size>>>(dm0->gpu(), dm1->gpu(), K, d_sigma, r_sigma);
}


void compute_vertex_map(const image<float>* dm, image<float3>* vm, intrinsics K)
{
    dim3 block_size(16, 16);
    dim3 grid_size;
    grid_size.x = divup(K.width, block_size.x);
    grid_size.y = divup(K.height, block_size.y);
    compute_vertex_kernel<<<grid_size, block_size>>>(dm->gpu(), vm->gpu(), K);
}


void compute_normal_map(const image<float3>* vm, image<float3>* nm, intrinsics K)
{
    dim3 block_size(16, 16);
    dim3 grid_size;
    grid_size.x = divup(K.width, block_size.x);
    grid_size.y = divup(K.height, block_size.y);
    compute_normal_kernel<<<grid_size, block_size>>>(vm->gpu(), nm->gpu(), K);
}


void compute_normal_radius_map(const image<float3>* vm, image<float4>* nm, intrinsics K)
{
    dim3 block_size(16, 16);
    dim3 grid_size;
    grid_size.x = divup(K.width, block_size.x);
    grid_size.y = divup(K.height, block_size.y);
    compute_normal_radius_kernel<<<grid_size, block_size>>>(vm->gpu(), nm->gpu(), K);
}


void reset_volume(volume<sdf32f_t>* vol)
{
    dim3 block_size(8, 8, 8);
    dim3 grid_size;
    grid_size.x = divup(vol->dimension.x, block_size.x);
    grid_size.y = divup(vol->dimension.y, block_size.y);
    grid_size.z = divup(vol->dimension.z, block_size.z);
    reset_volume_kernel<<<grid_size, block_size>>>(vol->gpu());
}
