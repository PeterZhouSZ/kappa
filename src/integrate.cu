#include "hip/hip_runtime.h"
#include <kappa/core.hpp>
#define INVALID_SURFEL 0xffffffff


__global__
void integrate_volume_kernel(volume<sdf32f_t> vol, image<float> dm, intrinsics K, mat4x4 T, float mu, float max_weight)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    if (x >= vol.dimension.x || y >= vol.dimension.y || z >= vol.dimension.z) return;

    float3 p ={(float)x, (float)y, (float)z};
    p = vol.offset + p * vol.voxel_size;
    float3 q = T * p;
    if (q.z <= 0.001f) return;

    int u = roundf((q.x / q.z) * K.fx + K.cx);
    int v = roundf((q.y / q.z) * K.fy + K.cy);
    if (u < 0 || u >= K.width || v < 0 || v >= K.height) return;

    float d = dm.data[u + v * K.width];
    if (d == 0.0f) return;

    float dist = d - q.z;
    if (dist <= -mu) return;

    float sigma_r = 0.6f;
    float max_rad_dist = sqrtf(K.width * K.width * 0.25f + K.height * K.height * 0.25f);
    float inv_r_sigma2 = -1.0 / (2.0f * sigma_r * sigma_r);
    float2 uv = {(float)(u - K.cx), (float)(v - K.cy)};
    float rad_dist = length(uv) / max_rad_dist;

    int i = x + y * vol.dimension.x + z * vol.dimension.x * vol.dimension.y;
    float ftt = fminf(1.0f, dist / mu);
    float wtt = __expf(rad_dist * rad_dist * inv_r_sigma2);
    float ft  = vol.data[i].tsdf;
    float wt  = vol.data[i].weight;
    vol.data[i].tsdf = (ft * wt + ftt * wtt) / (wt + wtt);
    vol.data[i].weight = fminf(wt + wtt, max_weight);
}


__global__
void match_surfel_kernel(image<uint32_t> im, image<uint32_t> mm, intrinsics K, mat4x4 T)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;
    if (u >= K.width || v >= K.height) return;

    int i = u + v * K.width;
    mm.data[i] = (im.data[i] == 0);
}


void integrate_volume(const volume<sdf32f_t>* vol, image<float>* dm, intrinsics K, mat4x4 T, float mu, float maxw)
{
    dim3 block_size(8, 8, 8);
    dim3 grid_size;
    grid_size.x = divup(vol->dimension.x, block_size.x);
    grid_size.y = divup(vol->dimension.y, block_size.y);
    grid_size.z = divup(vol->dimension.z, block_size.z);
    integrate_volume_kernel<<<grid_size, block_size>>>(vol->gpu(), dm->gpu(), K, T, mu, maxw);
}


void integrate_cloud(const cloud<surfel32f_t>* pc, image<float>* dm, image<uint32_t>* im, intrinsics K, mat4x4 T, float mu, float maxw)
{
    dim3 block_size(16, 16);
    dim3 grid_size;
    grid_size.x = divup(K.width, block_size.x);
    grid_size.y = divup(K.height, block_size.y);

    image<uint32_t> mm, sm;
    mm.allocate(K.width, K.height, DEVICE_CUDA);
    sm.allocate(K.width, K.height, DEVICE_CUDA);

    match_surfel_kernel<<<grid_size, block_size>>>(im->gpu(), mm.gpu(), K, T);
    sum_scan_cuda(mm.data, sm.data, K.width * K.height);

    sm.deallocate();
    mm.deallocate();
}
